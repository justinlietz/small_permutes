#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <cutt.h>

//
// Error checking wrapper for cutt
//
#define cuttCheck(stmt) do {                                 \
  cuttResult err = stmt;                            \
  if (err != CUTT_SUCCESS) {                          \
    fprintf(stderr, "%s in file %s, function %s\n", #stmt,__FILE__,__FUNCTION__); \
    exit(1); \
  }                                                  \
} while(0)

size_t idx(int i, int j, int k, int l, int* dims){
  size_t index = l*dims[0]*dims[1]*dims[2]
                +k*dims[0]*dims[1]
                +j*dims[0]
                +i;
  return index;
}

int equal_4tensors(double* data1, double*data2, int dims[4]){
  double diff;
  double tol = 1.e-10;
  for(int d3 = 0; d3 < dims[3]; d3++){
  for(int d2 = 0; d2 < dims[2]; d2++){
  for(int d1 = 0; d1 < dims[1]; d1++){
  for(int d0 = 0; d0 < dims[0]; d0++){
    size_t index = idx(d0,d1,d2,d3,dims);
    diff = fabs(data1[index] - data2[index]);
    if( diff > tol ){
      printf("diff at: %d,%d,%d,%d: %zu\n", d0,d1,d2,d3,index);
      return 0;
    }
  }
  }
  }
  }
  return 1;
}


void load_4tensor(double* data, int dims[4]){
  for(int d3 = 0; d3 < dims[3]; d3++){
  for(int d2 = 0; d2 < dims[2]; d2++){
  for(int d1 = 0; d1 < dims[1]; d1++){
  for(int d0 = 0; d0 < dims[0]; d0++){
    size_t index = idx(d0,d1,d2,d3,dims);
    /* printf("%d,%d,%d,%d: %zu\n", d0,d1,d2,d3,index); */
    data[index] = (double)d0*d1/d3+d2;
  }
  }
  }
  }
}

void print_4tensor(double* data, int dims[4]){
  for(int d3 = 0; d3 < dims[3]; d3++){
  for(int d2 = 0; d2 < dims[2]; d2++){
  for(int d1 = 0; d1 < dims[1]; d1++){
  for(int d0 = 0; d0 < dims[0]; d0++){
    size_t index = idx(d0,d1,d2,d3,dims);
    printf("%d,%d,%d,%d: %zu\n", d0,d1,d2,d3,index);
  }
  }
  }
  }
}

void transpose_4tensor(double* idata, double* odata, int dims[4], int perm[4]){
  int outdims[4];
  int outidx[4];
  int inidx[4];
  for( int i = 0; i<4; i++){
    outdims[i] = dims[ perm[i] ];
  }
  printf("outdims: %d %d %d %d\n",outdims[0], outdims[1], outdims[2], outdims[3]);
  for(int d3 = 0; d3 < dims[3]; d3++){
  for(int d2 = 0; d2 < dims[2]; d2++){
  for(int d1 = 0; d1 < dims[1]; d1++){
  for(int d0 = 0; d0 < dims[0]; d0++){
    inidx[0] = d0;
    inidx[1] = d1;
    inidx[2] = d2;
    inidx[3] = d3;
    for( int i = 0; i<4; i++){
      outidx[i] = inidx[ perm[i] ];
    }

    size_t iindex = idx(d0,d1,d2,d3,dims);
    size_t oindex = idx(outidx[0],outidx[1],outidx[2],outidx[3],outdims);
    odata[oindex] = idata[iindex];
  }
  }
  }
  }
}

int main() {

  // Four dimensional tensor
  // Transpose (31, 549, 2, 3) -> (3, 31, 2, 549)
  /* int dim[4] = {31, 549, 2, 3}; */
  /* int odim[4] = {3, 31, 2, 549}; */
  int dim[4] = {310, 5490, 2, 3};
  int odim[4] = {3, 310, 2, 5490};

  int permutation[4] = {3, 0, 2, 1};
  int nElems = 1;
  int nBytes;
  for(int i=0; i<4; i++){
    nElems = nElems*dim[i];
  }
  nBytes = sizeof(double)*nElems;
  printf("nBytes: %f Gb\n", nBytes/1.e9);

  /* .... input and output data is setup here ... */
  // double* idata : size product(dim)
  // double* odata : size product(dim)
  double* idata;
  double* ref_data;
  double* cutt_odata;
  double* d_idata;
  double* d_cutt_odata;

  idata = (double*)malloc(nBytes);
  ref_data = (double*)malloc(nBytes);
  cutt_odata = (double*)malloc(nBytes);
  hipMalloc((void**)&d_idata,nBytes);
  hipMalloc((void**)&d_cutt_odata,nBytes);


  load_4tensor(idata,dim);
  transpose_4tensor(idata,ref_data,dim,permutation);
  printf("nElems: %zu\n",nElems);

  hipMemcpy(d_idata,idata,nBytes,hipMemcpyHostToDevice);


  // Option 1: Create plan on NULL stream and choose implementation based on heuristics
  cuttHandle plan;
  cuttCheck(cuttPlan(&plan, 4, dim, permutation, sizeof(double), 0));

  // Option 2: Create plan on NULL stream and choose implementation based on performance measurements
  // cuttCheck(cuttPlanMeasure(&plan, 4, dim, permutation, sizeof(double), 0, idata, odata));

  // Execute plan
  cuttCheck(cuttExecute(plan, d_idata, d_cutt_odata));
  hipMemcpy(cutt_odata,d_cutt_odata,nBytes,hipMemcpyDeviceToHost);

  int ans;
  ans = equal_4tensors(ref_data, cutt_odata, odim);
  printf("ans: %d\n", ans);

  /* ... do stuff with your output and deallocate data ... */

  // Destroy plan
  cuttCheck(cuttDestroy(plan));

  return 0;
}


